#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#define DEVICE 1
#define nbSamples optixLaunchParams.frame.sampler

#include <optix_device.h>

#include "../common/LaunchParams.h"

  /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
  extern "C" __constant__ LaunchParams optixLaunchParams;

  
  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
  {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD()
  { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
  }
  
  //------------------------------------------------------------------------------
  // closest hit and anyhit programs for radiance-type rays.
  //
  // Note eventually we will have to create one pair of those for each
  // ray type and each geometry type we want to render; but this
  // simple example doesn't use any actual geometries yet, so we only
  // create a single, dummy, set of them (we do have to have at least
  // one group of them to set up the SBT)
  //------------------------------------------------------------------------------

  __device__ void swap(float &a, float &b) {
    float tmp = a;
    a = b;
    b = tmp;
  }
  __device__ bool inVolume(const VolumetricCube &data, const vec3f &pi){
        bool res = false;
        const vec3f min = data.center - data.size/2.0f;
        const vec3f max = data.center + data.size/2.0f;
        if(( pi.x <= max.x && pi.x >= min.x)&&
                ( pi.y <= max.y && pi.y >= min.y) &&
                ( pi.z <= max.z && pi.z >= min.z))
            res = true;
        return res;
  }
  
  __device__ void mip(){
      const VolumetricCube& data
       = (*(const sbtData*)optixGetSbtDataPointer()).volumeData;
     const int   primID = optixGetPrimitiveIndex();
     intersection_time time;
     time.tmin.uitmin = optixGetAttribute_0();
     time.tmax.uitmax = optixGetAttribute_1();
     //Gather information
     vec3f ro = optixGetWorldRayOrigin();
     vec3f rd = optixGetWorldRayDirection();
     vec3f& prd = *(vec3f*)getPRD<vec3f>();
     vec3f sizeP = vec3f(data.sizePixel.x, data.sizePixel.y, data.sizePixel.z);

     //Ray
     vec3f point_in = ro + time.tmin.ftmin * rd ;
     vec3f point_out = ro + time.tmax.ftmax * rd;
     vec3f ray_world = point_out - point_in;

     const float stepSize_current = norme(point_out - point_in) / nbSamples;
     vec3f step_vector_tex = normalize(ray_world) * stepSize_current;
     float current_ray_length = norme(ray_world);

     vec3f current_pos_tex = point_in;
     float current_max = 0.0f;
     float current_intensity = 0.0f;

     //MIP
     prd = vec3f(0.0f);
     while(current_ray_length > 0.0f){
        vec3f pos_tex = (current_pos_tex - data.center + data.size / 2.0f) / data.size;
        current_intensity = tex3D<float>(data.tex,pos_tex.x,pos_tex.y,pos_tex.z);


        if( current_intensity >= optixLaunchParams.frame.minIntensity && current_intensity <= optixLaunchParams.frame.maxIntensity){
            if( current_intensity > current_max )
                current_max = current_intensity;

            if( current_max == 1.0f){
                prd = vec3f(1.0f);
                break;
            }
            prd = vec3f(current_max);
        }
        current_pos_tex = current_pos_tex + step_vector_tex;
        current_ray_length -= stepSize_current;
     }
  }

  __device__ void minip(){
      const VolumetricCube& data
       = (*(const sbtData*)optixGetSbtDataPointer()).volumeData;
     intersection_time time;
     time.tmin.uitmin = optixGetAttribute_0();
     time.tmax.uitmax = optixGetAttribute_1();
     vec3f ro = optixGetWorldRayOrigin();
     vec3f rd = optixGetWorldRayDirection();
     vec3f& prd = *(vec3f*)getPRD<vec3f>();
     vec3f sizeP = vec3f(data.sizePixel.x, data.sizePixel.y, data.sizePixel.z);

     vec3f point_in = ro + time.tmin.ftmin * rd ;
     vec3f point_out = ro + time.tmax.ftmax * rd;
     vec3f ray_world = point_out - point_in;

     const float stepSize_current = norme(point_out - point_in) / nbSamples;
     vec3f step_vector_tex = normalize(ray_world) * stepSize_current;
     float current_ray_length = norme(ray_world);

     vec3f current_pos_tex = point_in;
     float current_min = 1.0f;
     float current_intensity = 0.0f;
     bool found = false;

     prd = vec3f(0.0f);
     while(current_ray_length > 0.0f){
        vec3f pos_tex = (current_pos_tex - data.center + data.size / 2.0f) / data.size;
        current_intensity = tex3D<float>(data.tex,pos_tex.x,pos_tex.y,pos_tex.z);

        if( current_intensity >= optixLaunchParams.frame.minIntensity && current_intensity <= optixLaunchParams.frame.maxIntensity){
            if( current_intensity < current_min ){
                current_min = current_intensity;
                found = true;
            }
        }
        current_pos_tex = current_pos_tex + step_vector_tex;
        current_ray_length -= stepSize_current;
     }
     if(found)
        prd = vec3f(current_min);
  }

  __device__ void meanip(){
      const VolumetricCube& data
       = (*(const sbtData*)optixGetSbtDataPointer()).volumeData;
     intersection_time time;
     time.tmin.uitmin = optixGetAttribute_0();
     time.tmax.uitmax = optixGetAttribute_1();
     vec3f ro = optixGetWorldRayOrigin();
     vec3f rd = optixGetWorldRayDirection();
     vec3f& prd = *(vec3f*)getPRD<vec3f>();
     vec3f sizeP = vec3f(data.sizePixel.x, data.sizePixel.y, data.sizePixel.z);

     vec3f point_in = ro + time.tmin.ftmin * rd ;
     vec3f point_out = ro + time.tmax.ftmax * rd;
     vec3f ray_world = point_out - point_in;

     const float stepSize_current = norme(point_out - point_in) / nbSamples;
     vec3f step_vector_tex = normalize(ray_world) * stepSize_current;
     float current_ray_length = norme(ray_world);

     vec3f current_pos_tex = point_in;
     float sum = 0.0f;
     int count = 0;

     prd = vec3f(0.0f);
     while(current_ray_length > 0.0f){
        vec3f pos_tex = (current_pos_tex - data.center + data.size / 2.0f) / data.size;
        float current_intensity = tex3D<float>(data.tex,pos_tex.x,pos_tex.y,pos_tex.z);

        if( current_intensity >= optixLaunchParams.frame.minIntensity && current_intensity <= optixLaunchParams.frame.maxIntensity){
            sum += current_intensity;
            count++;
        }
        current_pos_tex = current_pos_tex + step_vector_tex;
        current_ray_length -= stepSize_current;
     }
     if(count > 0)
        prd = vec3f(sum / count);
  }

  __device__ vec3f transferFunction(float intensity, float& alpha) {
      // Exemple simple : couleur en niveaux de gris, opacité croissante
      vec3f color = vec3f(intensity, intensity, intensity); // gris
      alpha = fminf(fmaxf((intensity - optixLaunchParams.frame.minIntensity) /
                          (optixLaunchParams.frame.maxIntensity - optixLaunchParams.frame.minIntensity), 0.0f), 1.0f);
      // On peut raffiner ici pour des couleurs plus complexes
      return color;
  }

  __device__ void dvr(){
      const VolumetricCube& data = (*(const sbtData*)optixGetSbtDataPointer()).volumeData;
      intersection_time time;
      time.tmin.uitmin = optixGetAttribute_0();
      time.tmax.uitmax = optixGetAttribute_1();
      vec3f ro = optixGetWorldRayOrigin();
      vec3f rd = optixGetWorldRayDirection();
      vec3f& prd = *(vec3f*)getPRD<vec3f>();
      vec3f point_in = ro + time.tmin.ftmin * rd;
      vec3f point_out = ro + time.tmax.ftmax * rd;
      vec3f ray_world = point_out - point_in;
      const float stepSize_current = norme(point_out - point_in) / nbSamples;
      vec3f step_vector_tex = normalize(ray_world) * stepSize_current;
      float current_ray_length = norme(ray_world);
      vec3f current_pos_tex = point_in;
      vec3f accum = vec3f(0.0f);
      float alpha_accum = 0.0f;
      while(current_ray_length > 0.0f && alpha_accum < 0.99f){
        vec3f pos_tex = (current_pos_tex - data.center + data.size / 2.0f) / data.size;
        float intensity = tex3D<float>(data.tex,pos_tex.x,pos_tex.y,pos_tex.z);
        if(intensity >= optixLaunchParams.frame.minIntensity && intensity <= optixLaunchParams.frame.maxIntensity){
            float alpha = 0.0f;
            vec3f color = transferFunction(intensity, alpha);
            accum = accum + (1.0f - alpha_accum) * color * alpha;
            alpha_accum += (1.0f - alpha_accum) * alpha;
        }
        current_pos_tex = current_pos_tex + step_vector_tex;
        current_ray_length -= stepSize_current;
      }
      prd = accum;
  }

  extern "C" __global__ void __closesthit__volume_radiance(){
      switch(optixLaunchParams.frame.renderType) {
        case 0: // MIP
          mip();
          break;
        case 1: // MINIP
          minip();
          break;
        case 2: // MEANIP
          meanip();
          break;
        case 3: // DVR
          dvr();
          break;
        default:
          mip();
          break;
      }
  }


  extern "C" __global__ void __anyhit__volume_radiance()
  {
  }


  extern "C" __global__ void __intersection__volume() {
      const VolumetricCube& sbtData
          = *(const VolumetricCube*)optixGetSbtDataPointer();
      vec3f ro = optixGetWorldRayOrigin();
      vec3f rayDir = optixGetWorldRayDirection();
      vec3f min, max;
      min = sbtData.center - sbtData.size / 2;
      max = sbtData.center + sbtData.size / 2;

      float tmin = (min.x - ro.x) / rayDir.x;
      float tmax = (max.x - ro.x) / rayDir.x;

      if (tmin > tmax) swap(tmin, tmax);

      float tymin = (min.y - ro.y) / rayDir.y;
      float tymax = (max.y - ro.y) / rayDir.y;

      if (tymin > tymax) swap(tymin, tymax);

      //Rayon en dehors du cube (normalement impossible )
      if ((tmin > tymax) || (tymin > tmax))

        return ;

      if (tymin > tmin)
          tmin = tymin;

      if (tymax < tmax)
          tmax = tymax;

      float tzmin = (min.z - ro.z) / rayDir.z;
      float tzmax = (max.z - ro.z) / rayDir.z;

      if (tzmin > tzmax) swap(tzmin, tzmax);

      //Rayon en dehors du cube (normalement impossible )
      if ((tmin > tzmax) || (tzmin > tmax))
        return ;

      if (tzmin > tmin)
          tmin = tzmin;

      if (tzmax < tmax)
          tmax = tzmax;
      if (tmin > tmax) swap(tmin, tmax);
      intersection_time time;

      time.tmin.ftmin = tmin;
      time.tmax.ftmax = tmax;
      optixReportIntersection(tmin, 1,time.tmin.uitmin, time.tmax.uitmax);
  }
